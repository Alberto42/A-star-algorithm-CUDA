#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <boost/program_options/options_description.hpp>
#include <boost/program_options/variables_map.hpp>
#include <boost/program_options.hpp>
#include <regex>
#include <cmath>

namespace po = boost::program_options;
using namespace std;

const int BLOCKS_COUNT = 1;
const int THREADS_PER_BLOCK_COUNT = 1;
const int THREADS_COUNT = BLOCKS_COUNT * THREADS_PER_BLOCK_COUNT;
const int MAX_SLIDES_COUNT = 25;
const int PRIORITY_QUEUE_SIZE = 100;
const int MAX_S_SIZE = 10;
const int INF = 1000000000;
const int H_SIZE = 1024; // It must be the power of 2
int slidesCount, slidesCountSqrt;

struct Vertex {
    int slides[MAX_SLIDES_COUNT];
    __host__ __device__

    Vertex(int slides[]) {
        memcpy(this->slides, slides, MAX_SLIDES_COUNT * sizeof(int));
    }

    __device__ Vertex() {}
    __device__ unsigned hashBase(int slidesCount, int base) {
        int result = 0;
        for(int i=0,p=1;i<slidesCount;i++,p=( p * base ) % H_SIZE) {
            result = (result + slides[i]*p) % H_SIZE;
        }
        return result;
    }
    __device__ int hash1(int slidesCount) {
        return this->hashBase(slidesCount, 30);
    }
    __device__ int hash2(int slidesCount) {
        int hash = this->hashBase(slidesCount, 29);
        hash = hash % 2 ? hash : (hash + 1) % H_SIZE;
        return hash;
    }
    __device__ int hash(int i, int slidesCount) {
        return (hash1(slidesCount) + i*hash2(slidesCount) ) % H_SIZE;
    }
};

__device__ bool vertexEqual(const Vertex &a, const Vertex &b,const int &slidesCount) {
    for (int i = 0; i < slidesCount; i++) {
        if (a.slides[i] != b.slides[i])
            return false;
    }
    return true;
}

struct State {
    Vertex node;
    int g, f, lock;
    State *prev;

    __device__ __host__ State():lock(1) {}

    __device__ __host__ State(int f):f(f), lock(1) {}

    __device__ __host__ State(int g, int f, Vertex node):g(g), f(f), node(node), lock(1) {}
    __device__ __host__ State& operator=(const State& that) {
        this->node = that.node;
        this->g = that.g;
        this->f = that.f;
        this->prev = that.prev;
        return *this;
    }
};

struct HashMap {
    State hashmap[H_SIZE];
    HashMap() {
        for(int i=0;i<H_SIZE;i++)
            hashmap[i].f = -1;
    }
    void insert(const State &s, int slidesCount) {
    }
    __device__ State* find(Vertex& v, int slidesCount) {
        for(int i=0;i<H_SIZE;i++) {
            int hash = v.hash(i,slidesCount);
            assert(0 <= hash && hash < H_SIZE);
            if (hashmap[i].f == -1 || vertexEqual(hashmap[hash].node,v, slidesCount))
                return &hashmap[hash];
        }
        assert(false);
        return nullptr;
    }
};
enum Version {
    sliding, pathfinding
};

struct Program_spec {
    Version version;
    ifstream in;
    ofstream out;
//    Program_spec(Version version, ifstream in, ofstream out):version(version),in(in),out(out){};
};

__device__ __host__ bool operator<(const State &a, const State &b) { return a.f < b.f; }

__device__ __host__ bool operator>(const State &a, const State &b) { return a.f > b.f; }

__device__ __host__ void swap(State &a, State &b) {
    State tmp = a;
    a = b;
    b = tmp;
}

__device__ __host__ void swap(int &a, int &b) {
    int tmp = a;
    a = b;
    b = tmp;
}
struct PriorityQueue {
    State A[PRIORITY_QUEUE_SIZE];
    int lock;

    __device__ PriorityQueue():lock(1) {}

    int heapSize = 0;

    __device__ __host__ int parent(int i) {
        return i / 2;
    }

    __device__ __host__ int left(int i) {
        return i * 2;
    }

    __device__ __host__ int right(int i) {
        return i * 2 + 1;
    }

    __device__ __host__ void maxHeapify(int i) {
        while(true) {
            int l = left(i);
            int r = right(i);
            int smallest;
            if (l <= heapSize && A[l] < A[i]) {
                smallest = l;
            } else {
                smallest = i;
            }
            if (r <= heapSize && A[r] < A[smallest])
                smallest = r;
            if (smallest != i) {
                swap(A[i], A[smallest]);
                i = smallest;
            } else {
                break;
            }
        }
    }

    __device__ __host__ void insert(State s) {
        assert(heapSize < PRIORITY_QUEUE_SIZE);
        heapSize++;
        A[heapSize] = s;
        int i = heapSize;
        while (i > 1 && A[parent(i)] > A[i]) {
            swap(A[i], A[parent(i)]);
            i = parent(i);
        }
    }

    __device__ State pop() {
        assert(heapSize > 0);
        State max = A[1];
        A[1] = A[heapSize];
        heapSize--;
        maxHeapify(1);
        return max;
    }

    __device__ bool empty() {
        return heapSize == 0;
    }

    __device__ State* top() {
        return (!this->empty()) ? A : nullptr;
    }
};

void parse_args(int argc, const char *argv[], Program_spec &program_spec) {
    po::options_description desc{"Options"};
    try {
        desc.add_options()
                ("version", po::value<std::string>(), "You have to specify version")
                ("input_data", po::value<std::string>())
                ("output_data", po::value<std::string>());
        po::variables_map vm;
        store(po::parse_command_line(argc, argv, desc), vm);
        po::notify(vm);
        if (vm.count("help")) {
            std::cout << desc << '\n';
            exit(0);
        }
        string version = vm["version"].as<string>();
        string input_file = vm["input_data"].as<string>();
        string output_file = vm["output_data"].as<string>();

        program_spec.in.open(input_file);
        program_spec.out.open(output_file);
        program_spec.version = version == "sliding" ? sliding : pathfinding;
    }
    catch (const po::error &ex) {
        std::cerr << ex.what() << '\n';
    }
    catch (...) {
        std::cerr << desc << '\n';
    }
}

void read_slides(ifstream &in, int *slides, int &len) {
    string s;
    getline(in, s);

    smatch m;
    regex e("_|[0-9]+");
    len = 0;
    while (regex_search(s, m, e)) {
        for (auto x:m) {
            slides[len++] = x == "_" ? 0 : stoi(x);
        }
        s = m.suffix().str();
    }
}

__device__ __host__ int f(const Vertex &a, const Vertex &b, int slidesCount, int slidesCountSqrt) {
    int pos[MAX_SLIDES_COUNT + 1];
    int sum = 0;
    for (int i = 0; i < slidesCount; i++) {
        int value = b.slides[i];
        if (value != 0) {
            assert(1 <= value && value <= slidesCount);
            pos[value] = i;
        }
    }
    for (int posA = 0; posA < slidesCount; posA++) {
        if (a.slides[posA] != 0) {
            int posB = pos[a.slides[posA]];
            int tmp1 = abs(posA % slidesCountSqrt - posB % slidesCountSqrt);
            int tmp2 = abs(posA / slidesCountSqrt - posB / slidesCountSqrt);
            sum += tmp1 + tmp2;
        }
    }
    return sum;
}

__device__ void expand(const State qi, State s[], int &sSize, const Vertex &target,int slidesCount, int
slidesCountSqrt) {
    int moves[] = {-1, 1, -slidesCountSqrt, slidesCountSqrt};
    const int movesCount = 4;
    int empty = -1;
    const int *slides = qi.node.slides;
    for (int i = 0; i < slidesCount; i++)
        if (slides[i] == 0) {
            empty = i;
            break;
        }
    if (empty == -1)
        assert(false);
    for (int i = 0; i < movesCount; i++) {
        int move = empty + moves[i];
        if (move < 0 || move >= slidesCount)
            continue;
        State sTmp = qi; // I hope slides is copied
        sTmp.g = qi.g + 1;

        swap(sTmp.node.slides[empty], sTmp.node.slides[move]);
        sTmp.f = f(sTmp.node, target, slidesCount, slidesCountSqrt);
        sTmp.prev = nullptr; //fixme
        sTmp.lock = 1;
        assert(sSize < MAX_S_SIZE);
        s[sSize++] = sTmp;
    }
}

__host__ int calcSlidesCountSqrt(int slidesCount) {
    int slidesCountSqrt;
    for (int i = 1; i < slidesCount; i++) {
        if (i * i == slidesCount) {
            slidesCountSqrt = i;
            break;
        }
        if (i == slidesCount - 1) {
            assert(false);
        }
    }
    return slidesCountSqrt;
}

__global__ void expandKernel(Vertex *start, Vertex *target, State *m, PriorityQueue *q, State *s, int *sSize,
                             int slidesCount, int slidesCountSqrt) {

    int id = threadIdx.x + blockIdx.x;
    sSize[id] = 0;
    if (q[id].empty()) {
        return;
    }
    State qi = q[id].pop();

    if (vertexEqual(qi.node,*target,slidesCount) ) {
        while (true) {
            int lock = atomicExch(&m->lock, 0);
            if (lock == 1) {
                if (qi.f < m->f) {
                    *m = qi;
                }
                lock = atomicExch(&m->lock, 1);
                assert(lock == 0);
                break;
            } else
                continue;
        }
    } else
        expand(qi, s + (id*MAX_S_SIZE), sSize[id], *target, slidesCount, slidesCountSqrt);
}
__global__ void checkIfTheEndKernel(State *m, PriorityQueue *q, int* result) {
    int id = threadIdx.x + blockIdx.x;
    State* t = q[id].top();
    if (t != nullptr && m->f > t->f) {
        atomicExch(result, 0); //fixme: Maybe atomic is not necessary
    }
}
__global__ void removeUselessStates(HashMap *h, State *t,int *sSize, int slidesCount) {
    int id = threadIdx.x + blockIdx.x;
    for(int i=id*MAX_S_SIZE;i < id*MAX_S_SIZE + sSize[id];i++) {
        assert(t[i].f != -1);
        State* tmp = h->find(t[i].node, slidesCount);
        if (tmp->g < t[i].g)
            t[i].f = -1;
    }
}
__global__ void insertNewStates(HashMap *h, State *t, int *sSize, PriorityQueue *q, int slidesCount) {
    int id = threadIdx.x + blockIdx.x;
    for(int i=id*MAX_S_SIZE;i < id*MAX_S_SIZE + sSize[id];i++) {
        if (t[i].f != -1 ) {
            while(true) {
                State *tmp = h->find(t[i].node, slidesCount);
                int lock = atomicExch(&t[i].lock, 0);
                if (lock) {
                    if (tmp->f == -1 || tmp->g > t[i].g) {
                        *tmp = t[i];
                        int hash = tmp->node.hash1(slidesCount) % THREADS_COUNT;
                        while(true) {
                            int lock = atomicExch(&q[hash].lock, 0);
                            if (lock) {
                                q[hash].insert(t[i]);
                                int lock = atomicExch(&q[hash].lock, 1);
                                assert(lock == 0);
                                break;
                            }
                        }
                    }
                    int lock = atomicExch(&t[i].lock, 1);
                    assert(lock == 0);
                    break;
                }
            }
        }
    }
}

void main2(int argc, const char *argv[]) {
    Program_spec result;
//    parse_args(argc, argv, result);
    result.in.open("slides/1.in");
    result.out.open("dupa");
    result.version = sliding;
    int slides[MAX_SLIDES_COUNT], slidesCount;

    read_slides(result.in, slides, slidesCount);
    slidesCountSqrt = calcSlidesCountSqrt(slidesCount);

    Vertex start(slides);
    read_slides(result.in, slides, slidesCount);
    Vertex target(slides);

    State m(INF);
    PriorityQueue q[THREADS_COUNT];
    HashMap h;
    int sSize[THREADS_COUNT], isTheEnd;
    q[0].insert(State(0, f(start, target, slidesCount, slidesCountSqrt), start));
    for(int i=0;i<THREADS_COUNT;i++) {
        sSize[i] = 0;
    }

    Vertex *devStart, *devTarget;
    State *devM, *devS;
    PriorityQueue *devQ;
    HashMap *devH;
    int *devSSize, *devIsTheEnd;

    hipMalloc(&devStart, sizeof(Vertex));
    hipMalloc(&devTarget, sizeof(Vertex));
    hipMalloc(&devM,sizeof(State));
    hipMalloc(&devQ,sizeof(PriorityQueue) * THREADS_COUNT);
    hipMalloc(&devS,sizeof(State) * THREADS_COUNT * MAX_S_SIZE);
    hipMalloc(&devSSize,sizeof(int) * THREADS_COUNT);
    hipMalloc(&devIsTheEnd, sizeof(int));
    hipMalloc(&devH, sizeof(HashMap));

    hipMemcpy(devStart, &start, sizeof(Vertex), hipMemcpyHostToDevice);
    hipMemcpy(devTarget, &target, sizeof(Vertex), hipMemcpyHostToDevice);
    hipMemcpy(devM, &m, sizeof(State), hipMemcpyHostToDevice);
    hipMemcpy(devQ, q, sizeof(PriorityQueue) * THREADS_COUNT, hipMemcpyHostToDevice);
    hipMemcpy(devSSize, sSize, sizeof(int) * THREADS_COUNT, hipMemcpyHostToDevice);
    hipMemcpy(devH, &h, sizeof(HashMap), hipMemcpyHostToDevice);


    while(true) {
        expandKernel << < BLOCKS_COUNT, THREADS_PER_BLOCK_COUNT >> > (devStart, devTarget, devM, devQ, devS, devSSize,
                slidesCount, slidesCountSqrt);

        isTheEnd = 1;
        hipMemcpy(devIsTheEnd, &isTheEnd, sizeof(int), hipMemcpyHostToDevice);
        checkIfTheEndKernel << < BLOCKS_COUNT, THREADS_PER_BLOCK_COUNT >> > (devM, devQ, devIsTheEnd);
        hipMemcpy(&isTheEnd, devIsTheEnd, sizeof(int), hipMemcpyDeviceToHost);
        if (isTheEnd) {
            break; //fixme
        }

        removeUselessStates <<<BLOCKS_COUNT, THREADS_PER_BLOCK_COUNT>>>(devH, devS, devSSize, slidesCount);

        insertNewStates <<<BLOCKS_COUNT, THREADS_PER_BLOCK_COUNT>>>(devH, devS, devSSize, devQ, slidesCount);
    }

    hipFree(devStart);
    hipFree(devTarget);
    hipFree(devM);
    hipFree(devQ);
    hipFree(devS);
    hipFree(devSSize);
    hipFree(devIsTheEnd);
    hipFree(devH);
}

int main(int argc, const char *argv[]) {
    main2(argc, argv);
}
