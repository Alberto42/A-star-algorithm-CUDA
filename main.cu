#include <iostream>
#include <fstream>
#include <boost/program_options/options_description.hpp>
#include <boost/program_options/variables_map.hpp>
#include <boost/program_options.hpp>
#include <regex>
#include <cmath>
#include <time.h>
#include "kernels/expandKernel.h"
#include "kernels/deduplicateKernel.h"
#include "kernels/insertNewStatesKernel.h"
#include "kernels/kernels.h"
#include "kernels/removeUselessStatesKernel.h"
#include "structures.h"

namespace po = boost::program_options;
using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

int slidesCount, slidesCountSqrt;

void parse_args(int argc, const char *argv[], Program_spec &program_spec) {
    po::options_description desc{"Options"};
    try {
        desc.add_options()
                ("version", po::value<std::string>(), "You have to specify version")
                ("input-data", po::value<std::string>())
                ("output-data", po::value<std::string>())
                ("device", po::value<int>()->default_value(1));
        po::variables_map vm;
        store(po::parse_command_line(argc, argv, desc), vm);
        po::notify(vm);
        if (vm.count("help")) {
            std::cout << desc << '\n';
            exit(0);
        }
        string version = vm["version"].as<string>();
        string input_file = vm["input-data"].as<string>();
        string output_file = vm["output-data"].as<string>();
        int device = vm["device"].as<int>();

        program_spec.in.open(input_file);
        program_spec.out.open(output_file);
        program_spec.version = version == "sliding" ? sliding : pathfinding;
        program_spec.device = device;
    }
    catch (const po::error &ex) {
        std::cerr << ex.what() << '\n';
    }
    catch (...) {
        std::cerr << desc << '\n';
    }
}

void read_slides(ifstream &in, int *slides, int &len) {
    string s;
    getline(in, s);

    smatch m;
    regex e("_|[0-9]+");
    len = 0;
    while (regex_search(s, m, e)) {
        for (auto x:m) {
            slides[len++] = x == "_" ? 0 : stoi(x);
        }
        s = m.suffix().str();
    }
    assert (len <= MAX_SLIDES_COUNT);
}

__host__ int calcSlidesCountSqrt(int slidesCount) {
    int slidesCountSqrt;
    for (int i = 1; i < slidesCount; i++) {
        if (i * i == slidesCount) {
            slidesCountSqrt = i;
            break;
        }
        if (i == slidesCount - 1) {
            assert(false);
        }
    }
    return slidesCountSqrt;
}

void main2(int argc, const char *argv[]) {
    Program_spec result;
    parse_args(argc, argv, result);
    int slides[MAX_SLIDES_COUNT], slidesCount;

    read_slides(result.in, slides, slidesCount);
    slidesCountSqrt = calcSlidesCountSqrt(slidesCount);

    Vertex start(slides);
    read_slides(result.in, slides, slidesCount);
    Vertex target(slides);

    State m(INF), qiCandidates[Q_CANDIDATES_COUNT];
    PriorityQueue q;
    int sSize[THREADS_COUNT], qiCandidatesCount=0, end=0;
    State startState = State(0, f(start, target, slidesCount, slidesCountSqrt), start);
    q.insert(startState);
    for(int i=0;i<THREADS_COUNT;i++) {
        sSize[i] = 0;
    }

    Vertex *devStart, *devTarget, *devPath;
    State *devM, *devS, *devT, *devQiCandidates;
    PriorityQueue *devQ;
    HashMap *devH;
    HashMapDeduplicate *devHD;
    int *devSSize, *devIsTheEnd, *devIsNotEmptyQueue, *devQiCandidatesCount, *devPathSize, *devEnd;

    gpuErrchk(hipSetDevice(result.device));
    gpuErrchk(hipMalloc(&devStart, sizeof(Vertex)));
    gpuErrchk(hipMalloc(&devTarget, sizeof(Vertex)));
    gpuErrchk(hipMalloc(&devM,sizeof(State)));
    gpuErrchk(hipMalloc(&devQ,sizeof(PriorityQueue) * THREADS_COUNT));
    gpuErrchk(hipMalloc(&devS,sizeof(State) * THREADS_COUNT * MAX_S_SIZE));
    gpuErrchk(hipMalloc(&devT,sizeof(State) * THREADS_COUNT * MAX_S_SIZE));
    gpuErrchk(hipMalloc(&devSSize,sizeof(int) * THREADS_COUNT));
    gpuErrchk(hipMalloc(&devIsTheEnd, sizeof(int)));
    gpuErrchk(hipMalloc(&devIsNotEmptyQueue, sizeof(int)));
    gpuErrchk(hipMalloc(&devQiCandidatesCount, sizeof(int)));
    gpuErrchk(hipMalloc(&devQiCandidates, sizeof(State) * Q_CANDIDATES_COUNT));
    gpuErrchk(hipMalloc(&devH, sizeof(HashMap)));
    gpuErrchk(hipMalloc(&devHD, sizeof(HashMapDeduplicate)));
    gpuErrchk(hipMalloc(&devPathSize, sizeof(int)));
    gpuErrchk(hipMalloc(&devEnd, sizeof(int)));

    gpuErrchk(hipMemcpy(devStart, &start, sizeof(Vertex), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(devTarget, &target, sizeof(Vertex), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(devM, &m, sizeof(State), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(devQ, &q, sizeof(PriorityQueue), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(devSSize, sSize, sizeof(int) * THREADS_COUNT, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(devQiCandidatesCount, &qiCandidatesCount, sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(devEnd, &end, sizeof(int), hipMemcpyHostToDevice));

    createHashmapKernel <<< BLOCKS_COUNT, THREADS_PER_BLOCK_COUNT >>> (devH, devStart, devTarget, slidesCount, slidesCountSqrt);

    hipEvent_t start_t, stop_t;
    gpuErrchk(hipEventCreate(&start_t));
    gpuErrchk(hipEventCreate(&stop_t));
    gpuErrchk(hipEventRecord(start_t, 0));

    while(true) {
        int isNotEmptyQueue = checkExistanceOfNotEmptyQueueHost(devQ,devIsNotEmptyQueue);
        if (!isNotEmptyQueue)
            break;

        expandKernel << < BLOCKS_COUNT, THREADS_PER_BLOCK_COUNT >> > (devStart, devTarget, devM, devQ, devS, devSSize,
                devQiCandidates, devQiCandidatesCount, slidesCount, slidesCountSqrt);
        improveMKernel <<< 1, 1 >>> (devM,devQiCandidates, devQiCandidatesCount);

        isNotEmptyQueue = checkExistanceOfNotEmptyQueueHost(devQ,devIsNotEmptyQueue);
        int isTheEnd = checkIfTheEndKernelHost(devM, devQ, devIsTheEnd);
        if (isTheEnd && isNotEmptyQueue)
            break;

        deduplicateKernelHost(devS,devSSize, devT, devHD, slidesCount);

        removeUselessStates <<<BLOCKS_COUNT, THREADS_PER_BLOCK_COUNT>>>(devH, devT, devSSize, slidesCount);

        insertNewStates <<<BLOCKS_COUNT, THREADS_PER_BLOCK_COUNT>>>(devH, devT, devSSize, devQ, devTarget,slidesCount,
                slidesCountSqrt, devEnd);
        gpuErrchk(hipMemcpy(&end, devEnd, sizeof(int), hipMemcpyDeviceToHost));
        if (end == 1)
            break;
    }

    gpuErrchk(hipEventRecord(stop_t, 0));

    gpuErrchk(hipMemcpy(&m, devM, sizeof(State), hipMemcpyDeviceToHost));

    float elapsedTime;
    gpuErrchk(hipEventElapsedTime(&elapsedTime, start_t, stop_t));


    result.out << elapsedTime << endl;
    if (m.f != INF) {
        gpuErrchk(hipMalloc(&devPath, sizeof(Vertex) * (m.g+10)));
        getPathKernel <<< 1, 1 >>> (devH, devM, devStart,slidesCount, devPath, devPathSize);
        int pathSize;
        gpuErrchk(hipMemcpy(&pathSize, devPathSize, sizeof(int), hipMemcpyDeviceToHost));
        Vertex* path = new Vertex[pathSize];
        gpuErrchk(hipMemcpy(path, devPath, sizeof(Vertex) * pathSize, hipMemcpyDeviceToHost));

        for(int i=pathSize-1;i>=0;i--)
            path[i].print(slidesCount,result.out);

        delete [] path;
        gpuErrchk(hipFree(devPath));
    }


    gpuErrchk(hipEventDestroy(start_t));
    gpuErrchk(hipEventDestroy(stop_t));
    gpuErrchk(hipFree(devStart));
    gpuErrchk(hipFree(devTarget));
    gpuErrchk(hipFree(devM));
    gpuErrchk(hipFree(devQ));
    gpuErrchk(hipFree(devS));
    gpuErrchk(hipFree(devSSize));
    gpuErrchk(hipFree(devIsTheEnd));
    gpuErrchk(hipFree(devH));
    gpuErrchk(hipFree(devHD));
    gpuErrchk(hipFree(devQiCandidates));
    gpuErrchk(hipFree(devQiCandidatesCount));
    gpuErrchk(hipFree(devPathSize));
    gpuErrchk(hipFree(devEnd));
}

int main(int argc, const char *argv[]) {
    main2(argc, argv);
}
